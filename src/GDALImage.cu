#include "hip/hip_runtime.h"
#include "GDALImage.h"
#include <iostream>

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <fcntl.h>
#include <assert.h>
#include <hipblas.h>
#include "hipError_t.h"
#include <errno.h>
#include <unistd.h>


/*! \brief Constructor
 *
 * @param filename a std::string with the raster image file name
 */

GDALImage::GDALImage(std::string filename, int band, int cacheSizeInGB)
{
    // open the file as dataset
    _poDataset = (GDALDataset *) GDALOpen(filename.c_str(), GA_ReadOnly );
    // if something is wrong, throw an exception
    // GDAL reports the error message
    if(!_poDataset)
        throw;

    // check the band info
    int count = _poDataset->GetRasterCount();
    if(band > count)
    {
        std::cout << "The desired band " << band << " is greated than " << count << " bands available";
        throw;
    }

    // get the desired band
    GDALRasterBand *poBand = _poDataset->GetRasterBand(band);
    if(!poBand)
        throw;

     // get the width(x), and height(y)
    _width = poBand->GetXSize();
    _height = poBand->GetYSize();



    char **papszOptions = NULL;
    // if cacheSizeInGB = 0, use default
    // else set the option
    if(cacheSizeInGB > 0)
        papszOptions = CSLSetNameValue( papszOptions,
            "CACHE_SIZE",
		    std::to_string(1024*1024*cacheSizeInGB).c_str());

    // space between two lines
	GIntBig pnLineSpace;
    // set up the virtual mem buffer
    _poBandVirtualMem =  GDALGetVirtualMemAuto(
        static_cast<GDALRasterBandH>(poBand),
		GF_Read,
		&_pixelSize,
		&pnLineSpace,
		papszOptions);

    // check it
    if(!_poBandVirtualMem)
        throw;

    // get the starting pointer
    _memPtr = CPLVirtualMemGetAddr(_poBandVirtualMem);

}





/// load a tile of data h_tile x w_tile from CPU (mmap) to GPU
/// @param dArray pointer for array in device memory
/// @param h_offset Down/Height offset
/// @param w_offset Across/Width offset
/// @param h_tile Down/Height tile size
/// @param w_tile Across/Width tile size
/// @param stream CUDA stream for copying
void GDALImage::loadToDevice(void *dArray, size_t h_offset, size_t w_offset, size_t h_tile, size_t w_tile, hipStream_t stream)
{
    size_t tileStartOffset = (h_offset*_width + w_offset)*_pixelSize;

    char * startPtr = (char *)_memPtr ;
    startPtr += tileStartOffset;

    // @note
    // We assume down/across directions as rows/cols. Therefore, SLC mmap and device array are both row major.
    // cuBlas assumes both source and target arrays are column major.
    // To use hipblasSetMatrix, we need to switch w_tile/h_tile for rows/cols
    // checkCudaErrors(hipblasSetMatrixAsync(w_tile, h_tile, sizeof(float2), startPtr, width, dArray, w_tile, stream));

    checkCudaErrors(hipMemcpy2DAsync(dArray, w_tile*_pixelSize, startPtr, _width*_pixelSize,
                                      w_tile*_pixelSize, h_tile, hipMemcpyHostToDevice,stream));
}

GDALImage::~GDALImage()
{
    // free the virtual memory
    CPLVirtualMemFree(_poBandVirtualMem),
    // free the GDAL Dataset, close the file
    delete _poDataset;
}

